#include "hip/hip_runtime.h"
//����EDR��GPU�����㷨
//zbw0046 3.22



#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include "ConstDefine.h"
#include "cudaKernel.h"
#include <assert.h>
#include"hip/device_functions.h"
#include "WinTimer.h"

#define CUDA_CALL(x) { const hipError_t a = (x); if (a!= hipSuccess) { printf("\nCUDA Error: %s(err_num=%d)\n", hipGetErrorString(a), a); hipDeviceReset(); assert(0);}}


/*
���м���1�����ģdp
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
iter: �ڼ���dp��λ��outputIdx����������ȫ���ڴ�λ�ã�tra1��tra2�������켣����ǰ�����빲���ڴ棻
*/
//__global__ void DPforward(const int iter, const int* outputIdx,const SPoint *tra1,const SPoint *tra2) {
//	SPoint p1 = tra1[threadIdx.x];
//	SPoint p2 = tra2[iter - threadIdx.x - 1]; //�������ڴ��Ǿۼ����ʵ���
//	bool subcost;
//	if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
//		subcost = 0;
//	}
//	else
//		subcost = 1;
//
//}

/*
SPoint�汾
case1���켣����С��512
���м���n��DP
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
queryTra[],candidateTra[][]:�켣
stateTableGPU[][]:��ÿ��candidate��state��
result[]:����ÿ��candidate��EDR���
�Ż�����
1���켣����share memory����
2��ֱ�Ӵ��ݹ켣����ʹ��ָ��
*/
__global__ void EDRDistance_1(SPoint *queryTra, SPoint **candidateTra,int candidateNum,int queryLength,int *candidateLength,int** stateTableGPU,int *result) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	if (blockID >= candidateNum) return;
	if ((threadID >= candidateLength[blockID]) && (threadID >= queryLength)) return;
	const int lenT = candidateLength[blockID];
	//int iterNum = queryLength;
	//if (lenT > queryLength)
	//	iterNum = lenT;
	const int iterNum = queryLength + lenT - 1;
	__shared__ int state[2][MAXTHREAD]; //���ڴ洢ǰ���εĽ��
	state[0][0] = 0;
	state[1][0] = 1;
	state[1][1] = 1;
	//�������켣���򣬱�֤��һ���ȵڶ�����
	//���Ȱѹ켣���ڹ����ڴ���
	__shared__ SPoint queryTraS[MAXTHREAD];
	__shared__ SPoint traData[MAXTHREAD];
	if (threadID < lenT) {
		traData[threadID] = candidateTra[blockID][threadID];
	}
	if (threadID < queryLength) {
		queryTraS[threadID] = queryTra[threadID];
	}
	const SPoint *tra1, *tra2; //��֤tra1��tra2��
	int len1, len2;
	if (lenT >= queryLength) {
		tra1 = queryTraS;
		tra2 = traData;
		len1 = queryLength;
		len2 = lenT;
	}
	else
	{
		tra1 = traData;
		tra2 = queryTraS;
		len1 = lenT;
		len2 = queryLength;
	}

	int myState;
	for (int i = 0; i <= iterNum - 1; i++) {//��i��dp
		if (i < len1 - 1) {
			if (threadID <= i) {
				SPoint p1 = tra1[threadID];
				SPoint p2 = tra2[i - threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
					subcost = 0;
				}
				else
					subcost = 1;
				int state_ismatch = state[0][threadID] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID+1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_ismatch;
			}
		}
		else if (i > iterNum - len1) {
			if (threadID <= iterNum - i - 1) {
				SPoint p1 = tra1[threadID+len1-(iterNum-i)];
				SPoint p2 = tra2[len2-1-threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
					subcost = 0;
				}
				else
					subcost = 1;
				int state_ismatch = state[0][threadID+1] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID + 1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_ismatch;
			}
		}
		else
		{
			if (threadID < len1) {
				SPoint p1 = tra1[threadID];
				SPoint p2 = tra2[i-threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
					subcost = 0;
				}
				else
					subcost = 1;
				int state_ismatch = state[0][threadID] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID + 1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_ismatch;
			}
		}
		//дmyState��share�ڴ�,ckecked
		int startidx;
		//���Ƚ�������д��ȫ���ڴ棬ȫд
		//startidx�Ǿɵ�����Ӧ����ȫ���ڴ��е�ַ����i-2����
		//����Ӧд��ȫ���ڴ����ʼλ��

		if (i-2 < len1 - 2) {
			startidx = (i-2 + 2)*(i-2 + 3) / 2;
			if (threadID <= i) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}
		else if (i-2 >= iterNum - len1) {
			startidx = (len1 + 1)*(len2 + 1) - (iterNum - (i-2))*(iterNum - (i-2) + 1) / 2;
			if (threadID <= iterNum - i + 1 ) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}
		else
		{
			startidx = (len1 + 1)*((i - 2) - (len1 - 2)) + len1*(len1 + 1) / 2;
			if (threadID <= len1) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}

		//�ƶ������ݵ�������
		state[0][threadID] = state[1][threadID];
		//д��������
		if (i < len1-1) {
			if (threadID <= i)
				state[1][threadID + 1] = myState;
			if (threadID == 0) {
				state[1][0] = i + 2;
				state[1][i + 2] = i + 2;
			}
		}
		else if (i >= iterNum - len1) {
			if (threadID <= iterNum - i - 1)
				state[1][threadID] = myState;
		}
		else
		{
			if (threadID < len1)
				state[1][threadID + 1] = myState;
			if (threadID == 0) {
				state[1][0] = i + 2;
			}
		}
		__syncthreads();
	}
	//�����������һ�μ���һ�����ɽ���0��ɵ�
	if (threadID == 0)
		result[blockID] = myState;
}
/*
//�Ȱ����ܷ���һ��SMִ��һ��DP�����������ٷֱ��������kernel
//constructing...
���Ż���
1��queryTra��queryLength����candidateLength����ͨ����ֵ�ķ�ʽֱ�Ӵ��ݵ�SM�ļĴ���������ȫ���ڴ��ʹ��

*/
int handleEDRdistance(SPoint *queryTra, SPoint **candidateTra, int candidateNum, int queryLength, int *candidateLength,int *result) {
	MyTimer time1;
	time1.start();

	int** stateTableGPU=NULL;
	//��GPU��Ϊ״̬������ڴ�
	int** temp=NULL;
	temp = (int**)malloc(sizeof(int*)*candidateNum);
	for (int i = 0; i <= candidateNum - 1; i++) {
		CUDA_CALL(hipMalloc((void**)&temp[i], sizeof(int)*(candidateLength[i] + 1)*(queryLength + 1)));
	}
	CUDA_CALL(hipMalloc((void***)&stateTableGPU, sizeof(int*)*candidateNum));
	CUDA_CALL(hipMemcpy(stateTableGPU, temp, candidateNum*sizeof(int*), hipMemcpyHostToDevice));

	//Ϊ�洢�Ĺ켣��Ϣ�����ڴ�
	SPoint *queryTraGPU=NULL, **candidateTraGPU=NULL;
	int *candidateLengthGPU=NULL, *resultGPU=NULL;
	CUDA_CALL(hipMalloc((void**)&queryTraGPU, sizeof(SPoint)*queryLength));
	CUDA_CALL(hipMalloc((void**)&candidateLengthGPU, sizeof(int)*candidateNum));
	//CUDA_CALL(hipMalloc((void**)&resultGPU, sizeof(int)*candidateNum));

	SPoint **tempS = (SPoint**)malloc(sizeof(SPoint*)*candidateNum);
	for (int i = 0; i <= candidateNum - 1; i++) {
		CUDA_CALL(hipMalloc((void**)&tempS[i], sizeof(SPoint)*candidateLength[i]));
		
	}
	CUDA_CALL(hipMalloc((void***)&candidateTraGPU, sizeof(SPoint*)*candidateNum));
	CUDA_CALL(hipMemcpy(candidateTraGPU, tempS, candidateNum*sizeof(SPoint*), hipMemcpyHostToDevice));

	//���ͨ���������ķ������ݹ켣�����Ҫ��켣�����洢
	//��GPU���ݹ켣��Ϣ
	CUDA_CALL(hipMemcpy(queryTraGPU, queryTra, queryLength*sizeof(SPoint), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(candidateLengthGPU, candidateLength, candidateNum*sizeof(int), hipMemcpyHostToDevice));
	
	for (int i = 0; i <= candidateNum - 1; i++) {
		CUDA_CALL(hipMemcpy(tempS[i], candidateTra[i], candidateLength[i] * sizeof(SPoint), hipMemcpyHostToDevice));
	}
	//for (int i = 0; i <= candidateNum - 1;i++)
	//	CUDA_CALL(hipMemcpy(candidateTraGPU[i], candidateTra[i], candidateLength[i]*sizeof(SPoint), hipMemcpyHostToDevice));
	CUDA_CALL(hipHostAlloc((void**)&result, candidateNum*sizeof(int), hipHostMallocWriteCombined | hipHostMallocMapped));
	CUDA_CALL(hipHostGetDevicePointer(&resultGPU, result, 0));
	time1.stop();
	std::cout << time1.elapse() << std::endl;
	time1.start();
	//ִ��kernel
	EDRDistance_1 <<<candidateNum, MAXTHREAD >>>(queryTraGPU, candidateTraGPU, candidateNum, queryLength, candidateLengthGPU, stateTableGPU, resultGPU);

	//ȡ���
	//result = (int*)malloc(candidateNum*sizeof(int));
	//CUDA_CALL(hipMemcpy(result, resultGPU, candidateNum*sizeof(int), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
//	for (int j = 0; j <= candidateNum - 1;j++)
//		std::cout << result[j] << std::endl;

	//free GPU!!!!!
	time1.stop();
	std::cout << time1.elapse() << std::endl;
	return 0;

}

