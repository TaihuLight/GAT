#include "hip/hip_runtime.h"
//����EDR��GPU�����㷨
//zbw0046 3.22



#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include "ConstDefine.h"
#include "cudaKernel.h"
#include <assert.h>
#include"hip/device_functions.h"
#include "WinTimer.h"

#define CUDA_CALL(x) { const hipError_t a = (x); if (a!= hipSuccess) { printf("\nCUDA Error: %s(err_num=%d)\n", hipGetErrorString(a), a); hipDeviceReset(); assert(0);}}


/*
���м���1�����ģdp
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
iter: �ڼ���dp��λ��outputIdx����������ȫ���ڴ�λ�ã�tra1��tra2�������켣����ǰ�����빲���ڴ棻
*/
//__global__ void DPforward(const int iter, const int* outputIdx,const SPoint *tra1,const SPoint *tra2) {
//	SPoint p1 = tra1[threadIdx.x];
//	SPoint p2 = tra2[iter - threadIdx.x - 1]; //�������ڴ��Ǿۼ����ʵ���
//	bool subcost;
//	if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
//		subcost = 0;
//	}
//	else
//		subcost = 1;
//
//}

/*
SPoint�汾
case1���켣����С��512
���м���n��DP
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
queryTra[],candidateTra[][]:�켣
stateTableGPU[][]:��ÿ��candidate��state��
result[]:����ÿ��candidate��EDR���
�Ż�����
1���켣����share memory����
2��ֱ�Ӵ��ݹ켣����ʹ��ָ��
*/
__global__ void EDRDistance_1(SPoint *queryTra, SPoint **candidateTra,int candidateNum,int queryLength,int *candidateLength,int** stateTableGPU,int *result) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	if (blockID >= candidateNum) return;
	if ((threadID >= candidateLength[blockID]) && (threadID >= queryLength)) return;
	const int lenT = candidateLength[blockID];
	//int iterNum = queryLength;
	//if (lenT > queryLength)
	//	iterNum = lenT;
	const int iterNum = queryLength + lenT - 1;
	__shared__ int state[2][MAXTHREAD]; //���ڴ洢ǰ���εĽ��
	state[0][0] = 0;
	state[1][0] = 1;
	state[1][1] = 1;
	//�������켣���򣬱�֤��һ���ȵڶ�����
	//���Ȱѹ켣���ڹ����ڴ���
	__shared__ SPoint queryTraS[MAXTHREAD];
	__shared__ SPoint traData[MAXTHREAD];
	if (threadID < lenT) {
		traData[threadID] = candidateTra[blockID][threadID];
	}
	if (threadID < queryLength) {
		queryTraS[threadID] = queryTra[threadID];
	}
	const SPoint *tra1, *tra2; //��֤tra1��tra2��
	int len1, len2;
	if (lenT >= queryLength) {
		tra1 = queryTraS;
		tra2 = traData;
		len1 = queryLength;
		len2 = lenT;
	}
	else
	{
		tra1 = traData;
		tra2 = queryTraS;
		len1 = lenT;
		len2 = queryLength;
	}

	int myState;
	for (int i = 0; i <= iterNum - 1; i++) {//��i��dp
		if (i < len1 - 1) {
			if (threadID <= i) {
				SPoint p1 = tra1[threadID];
				SPoint p2 = tra2[i - threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				//if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
				//	subcost = 0;
				//}
				//else
				//	subcost = 1;
				subcost = !(((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON);
				int state_ismatch = state[0][threadID] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID+1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_up;
				//ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
				//myState = (state_ismatch < state_up) * state_ismatch + (state_left < state_up) * state_up + (state_left >= state_up) * state_left;
				
			}
		}
		else if (i > iterNum - len1) {
			if (threadID <= iterNum - i - 1) {
				SPoint p1 = tra1[threadID+len1-(iterNum-i)];
				SPoint p2 = tra2[len2-1-threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
					subcost = 0;
				}
				else
					subcost = 1;
				int state_ismatch = state[0][threadID+1] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID + 1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_up;
			}
		}
		else
		{
			if (threadID < len1) {
				SPoint p1 = tra1[threadID];
				SPoint p2 = tra2[i-threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
					subcost = 0;
				}
				else
					subcost = 1;
				int state_ismatch = state[0][threadID] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID + 1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_up;
			}
		}
		//дmyState��share�ڴ�,ckecked
		int startidx;
		//���Ƚ�������д��ȫ���ڴ棬ȫд
		//startidx�Ǿɵ�����Ӧ����ȫ���ڴ��е�ַ����i-2����
		//����Ӧд��ȫ���ڴ����ʼλ��

		if (i-2 < len1 - 2) {
			startidx = (i-2 + 2)*(i-2 + 3) / 2;
			if (threadID <= i) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}
		else if (i-2 >= iterNum - len1) {
			startidx = (len1 + 1)*(len2 + 1) - (iterNum - (i-2))*(iterNum - (i-2) + 1) / 2;
			if (threadID <= iterNum - i + 1 ) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}
		else
		{
			startidx = (len1 + 1)*((i - 2) - (len1 - 2)) + len1*(len1 + 1) / 2;
			if (threadID <= len1) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}

		//�ƶ������ݵ�������
		state[0][threadID] = state[1][threadID];
		//д��������
		if (i < len1-1) {
			if (threadID <= i)
				state[1][threadID + 1] = myState;
			if (threadID == 0) {
				state[1][0] = i + 2;
				state[1][i + 2] = i + 2;
			}
		}
		else if (i >= iterNum - len1) {
			if (threadID <= iterNum - i - 1)
				state[1][threadID] = myState;
		}
		else
		{
			if (threadID < len1)
				state[1][threadID + 1] = myState;
			if (threadID == 0) {
				state[1][0] = i + 2;
			}
		}
		__syncthreads();
	}
	//�����������һ�μ���һ�����ɽ���0��ɵ�
	if (threadID == 0)
		result[blockID] = myState;
}


//__global__ void testSharedMemory()
//{
//	__shared__ SPoint queryTraS[MAXLENGTH];
//	__shared__ SPoint traData[MAXLENGTH];
//	__shared__ SPoint traData2[MAXLENGTH];
//	SPoint s;
//	s.x = 4;
//	s.y = 5;
//	traData[1535] = s;
//	queryTraS[1535] = s;
//	traData2[1535] = s;
//}


/*
SPoint�汾
ͬʱ�������ɸ�query��EDR��������һ��EDR����Ϊ��λ��ÿ��block����һ��EDR��thread����һ��б����state�Ĳ��м��㡣
case1���켣���ȿɳ���512������ѭ���������512�ġ�
���м���n��DP
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
queryTaskNum:�ܹ��м���EDR��������
queryTaskInfo[]��ÿ��task��Ӧ��qID��candidateID��Ϣ����struct�洢
queryTra[],candidateTra[]:�켣���ݣ�candidateTra��֤���ڲ��켣���ظ�
queryTraOffset[],candidateTraOffset[]:ÿ���켣��offset��candidateTra��֤���ڲ��켣���ظ�
queryLength[],candidateLength[]:ÿ���켣�ĳ��ȣ���ʵoffset������ǳ��ȣ�����idx������Ķ�Ӧ
����candidateLength[id]�ǵ�id��candidate Traj�ĳ���
stateTableGPU[][]:��ÿ��candidate��state��
result[]:����ÿ��candidate��EDR���
�Ż�����
1���켣����share memory����
2��ֱ�Ӵ��ݹ켣����ʹ��ָ��
*/



__global__ void EDRDistance_Batch(int queryTaskNum, TaskInfoTableForSimilarity* taskInfoTable, SPoint *queryTra, int* queryTraOffset, SPoint** candidateTraOffsets, int* queryLength, int *candidateLength, int *result) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	if (blockID >= queryTaskNum) return;
	int thisQueryID = taskInfoTable[blockID].qID;
	int thisQueryLength = queryLength[thisQueryID];
	if ((threadID >= candidateLength[blockID]) && (threadID >= thisQueryLength)) return;
	const int lenT = candidateLength[blockID];
	//int iterNum = queryLength;
	//if (lenT > queryLength)
	//	iterNum = lenT;
	const int iterNum = thisQueryLength + lenT - 1;
	__shared__ int state[2][MAXLENGTH]; //���ڴ洢ǰ���εĽ����ռ��8KB��
	state[0][0] = 0;
	state[1][0] = 1;
	state[1][1] = 1;
	//�������켣���򣬱�֤��һ���ȵڶ�����
	//���Ȱѹ켣���ڹ����ڴ���
	//����������share memory�Ƿ��õ����⣬����д����64KB��Ȼ��K80�ƺ���512KB
	//�����64KB�Ļ���ÿ���켣�1024���㣨�����켣��ռ��24KB��
	__shared__ SPoint queryTraS[MAXLENGTH];
	__shared__ SPoint traData[MAXLENGTH];

	for (int i = 0; i <= lenT - 1;i+=MAXTHREAD)
	{
		if(threadID+i<lenT)
		{
			traData[threadID + i] = SPoint(candidateTraOffsets[blockID][threadID + i]);
		}
	}

	SPoint* queryTraBaseAddr = queryTra + queryTraOffset[thisQueryID];
	for (int i = 0; i <= thisQueryLength - 1;i+=MAXTHREAD)
	{
		if(threadID+i<thisQueryLength)
		{
			queryTraS[threadID + i] = *(queryTraBaseAddr + threadID + i);
		}
	}

	const SPoint *tra1, *tra2; //��֤tra1��tra2��
	int len1, len2;
	if (lenT >= thisQueryLength) {
		tra1 = queryTraS;
		tra2 = traData;
		len1 = thisQueryLength;
		len2 = lenT;
	}
	else
	{
		tra1 = traData;
		tra2 = queryTraS;
		len1 = lenT;
		len2 = thisQueryLength;
	}

	int myState[5];
	int nodeID;
	for (int i = 0; i <= iterNum - 1; i++) {//��i��dp
		if (i < len1 - 1) {
			for (int startIdx = 0; startIdx <= i; startIdx += MAXTHREAD) {
				nodeID = startIdx + threadID;
				if (nodeID <= i) {
					SPoint p1 = tra1[nodeID];
					SPoint p2 = tra2[i - nodeID]; //�������ڴ��Ǿۼ����ʵ���
					bool subcost;
					if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
						subcost = 0;
					}
					else
						subcost = 1;
					//subcost = !(((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON);
					int state_ismatch = state[0][nodeID] + subcost;
					int state_up = state[1][nodeID] + 1;
					int state_left = state[1][nodeID + 1] + 1;
					bool c1 = ((state_ismatch < state_up) && (state_ismatch < state_left));
					bool c2 = ((state_left < state_up) && ((state_left < state_ismatch)));
					//ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
					myState[nodeID / MAXTHREAD] = c1 * state_ismatch + c2 * state_left + !(c1 || c2) * state_up;
					//if ((state_ismatch < state_up) && (state_ismatch < state_left))
					//	myState[nodeID/MAXTHREAD] = state_ismatch;
					//else if ((state_left < state_up) && ((state_left < state_ismatch)))
					//	myState[nodeID / MAXTHREAD] = state_left;
					//else
					//	myState[nodeID / MAXTHREAD] = state_up;
					////ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
					//myState[nodeID / MAXTHREAD] = (state_ismatch < state_up) && (state_ismatch < state_left) * state_ismatch + ((state_left < state_up) && ((state_left < state_ismatch))) * state_left + !(((state_ismatch < state_up) && (state_ismatch < state_left))||(((state_left < state_up) && ((state_left < state_ismatch))))) * state_up;
				}
			}
		}
		else if (i > iterNum - len1) {
			for (int startIdx = 0; startIdx <= iterNum - i - 1; startIdx += MAXTHREAD) {
				nodeID = startIdx + threadID;
				if (nodeID <= iterNum - i - 1) {
					SPoint p1 = tra1[nodeID + len1 - (iterNum - i)];
					SPoint p2 = tra2[len2 - 1 - nodeID]; //�������ڴ��Ǿۼ����ʵ���
					bool subcost;
					if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
						subcost = 0;
					}
					else
						subcost = 1;
					int state_ismatch = state[0][nodeID + 1] + subcost;
					int state_up = state[1][nodeID] + 1;
					int state_left = state[1][nodeID + 1] + 1;
					//if (state_ismatch < state_up)
					//	myState[nodeID / MAXTHREAD] = state_ismatch;
					//else if (state_left < state_up)
					//	myState[nodeID / MAXTHREAD] = state_left;
					//else
					//	myState[nodeID / MAXTHREAD] = state_up;
					bool c1 = ((state_ismatch < state_up) && (state_ismatch < state_left));
					bool c2 = ((state_left < state_up) && ((state_left < state_ismatch)));
					//ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
					myState[nodeID / MAXTHREAD] = c1 * state_ismatch + c2 * state_left + !(c1 || c2) * state_up;
				}
			}
		}
		else
		{
			for (int startIdx = 0; startIdx <= len1; startIdx += MAXTHREAD) {
				nodeID = startIdx + threadID;
				if (nodeID <= len1) {
					SPoint p1 = tra1[nodeID];
					SPoint p2 = tra2[i - nodeID]; //�������ڴ��Ǿۼ����ʵ���
					bool subcost;
					if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
						subcost = 0;
					}
					else
						subcost = 1;
					int state_ismatch = state[0][nodeID] + subcost;
					int state_up = state[1][nodeID] + 1;
					int state_left = state[1][nodeID + 1] + 1;
					//if (state_ismatch < state_up)
					//	myState[nodeID / MAXTHREAD] = state_ismatch;
					//else if (state_left < state_up)
					//	myState[nodeID / MAXTHREAD] = state_left;
					//else
					//	myState[nodeID / MAXTHREAD] = state_up;
					bool c1 = ((state_ismatch < state_up) && (state_ismatch < state_left));
					bool c2 = ((state_left < state_up) && ((state_left < state_ismatch)));
					//ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
					myState[nodeID / MAXTHREAD] = c1 * state_ismatch + c2 * state_left + !(c1 || c2) * state_up;
				}
			}
		}
		//дmyState��share�ڴ�,ckecked
		
		//int startidx;
		////���Ƚ�������д��ȫ���ڴ棬ȫд
		////startidx�Ǿɵ�����Ӧ����ȫ���ڴ��е�ַ����i-2����
		////����Ӧд��ȫ���ڴ����ʼλ��
		//// 7.2 ���֣��ƺ�stateTableGPU������д������������
		////////

		//if (i - 2 < len1 - 2) {
		//	startidx = (i - 2 + 2)*(i - 2 + 3) / 2;
		//	for (int Idx = 0; Idx <= i; Idx += MAXTHREAD) {
		//		//if (threadID <= i) {
		//		if(Idx + threadID <= i){
		//			//stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
		//			stateTableGPU[blockID][Idx + threadID + startidx] = state[0][threadID + Idx];
		//		}
		//	}
		//}
		//else if (i - 2 >= iterNum - len1) {
		//	startidx = (len1 + 1)*(len2 + 1) - (iterNum - (i - 2))*(iterNum - (i - 2) + 1) / 2;
		//	for (int Idx = 0; Idx <= iterNum - i + 1; Idx += MAXTHREAD) {
		//		//if (threadID <= iterNum - i + 1) {
		//		if (threadID + Idx <= iterNum - i + 1) {
		//			//stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
		//			stateTableGPU[blockID][Idx + threadID + startidx] = state[0][Idx + threadID];
		//		}
		//	}
		//}
		//else
		//{
		//	startidx = (len1 + 1)*((i - 2) - (len1 - 2)) + len1*(len1 + 1) / 2;
		//	for (int Idx = 0; Idx <= len1; Idx += MAXTHREAD) {
		//		//if (threadID <= len1) {
		//		if (threadID + Idx <= len1) {
		//			// stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
		//			stateTableGPU[blockID][Idx + threadID + startidx] = state[0][Idx + threadID];
		//		}
		//	}
		//}

		//�ƶ������ݵ�������
		for (int Idx = 0; Idx < MAXLENGTH;Idx+=MAXTHREAD)
		{
			state[0][threadID+Idx] = state[1][threadID+Idx];
		}
		//state[0][threadID] = state[1][threadID];

		//д��������
		if (i < len1 - 1) {
			//if (threadID <= i)
			//	state[1][threadID + 1] = myState;
			//if (threadID == 0) {
			//	state[1][0] = i + 2;
			//	state[1][i + 2] = i + 2;
			//}
			for (int Idx = 0; Idx <= i; Idx += MAXTHREAD) {
				if (threadID + Idx <= i)
					state[1][Idx + threadID + 1] = myState[Idx/MAXTHREAD];
			}
			if (threadID == 0) {
				state[1][0] = i + 2;
				state[1][i + 2] = i + 2;
			}
		}
		else if (i >= iterNum - len1) {
			//if (threadID <= iterNum - i - 1)
			//	state[1][threadID] = myState;
			for (int Idx = 0; Idx <= iterNum - i - 1; Idx += MAXTHREAD) {
				if (threadID + Idx <= iterNum - i - 1)
					state[1][threadID + Idx] = myState[Idx / MAXTHREAD];
			}
		}
		else
		{
			//if (threadID < len1)
			//	state[1][threadID + 1] = myState;
			//if (threadID == 0) {
			//	state[1][0] = i + 2;
			//}
			for (int Idx = 0; Idx <= len1; Idx += MAXTHREAD) {
				if (threadID + Idx < len1)
					state[1][Idx + threadID + 1] = myState[Idx / MAXTHREAD];
			}
			if (threadID == 0) {
				state[1][0] = i + 2;
			}
		}
		__syncthreads();
	}
	//�����������һ�μ���һ�����ɽ���0��ɵ�
	if (threadID == 0 && blockID < queryTaskNum)
		result[blockID] = myState[0];
}

int EDRDistance_Batch_Handler(int queryTaskNum, TaskInfoTableForSimilarity* taskInfoTable, SPoint *queryTra, int* queryTraOffset, SPoint** candidateTraOffsets, int* queryLength, int *candidateLength, int *result, hipStream_t *stream)
{
	EDRDistance_Batch <<<queryTaskNum, MAXTHREAD,0 , *stream >>>(queryTaskNum, taskInfoTable, queryTra, queryTraOffset, candidateTraOffsets, queryLength, candidateLength, result);
	return 0;
}


/*
//�Ȱ����ܷ���һ��SMִ��һ��DP�����������ٷֱ��������kernel
//constructing...
���Ż���
1��queryTra��queryLength����candidateLength����ͨ����ֵ�ķ�ʽֱ�Ӵ��ݵ�SM�ļĴ���������ȫ���ڴ��ʹ��

*/
int handleEDRdistance(SPoint *queryTra, SPoint **candidateTra, int candidateNum, int queryLength, int *candidateLength,int *result) {
	MyTimer time1;
	time1.start();

	int** stateTableGPU=NULL;
	//��GPU��Ϊ״̬������ڴ�
	int** temp=NULL;
	temp = (int**)malloc(sizeof(int*)*candidateNum);
	for (int i = 0; i <= candidateNum - 1; i++) {
		CUDA_CALL(hipMalloc((void**)&temp[i], sizeof(int)*(candidateLength[i] + 1)*(queryLength + 1)));
	}
	CUDA_CALL(hipMalloc((void***)&stateTableGPU, sizeof(int*)*candidateNum));
	CUDA_CALL(hipMemcpy(stateTableGPU, temp, candidateNum*sizeof(int*), hipMemcpyHostToDevice));

	//Ϊ�洢�Ĺ켣��Ϣ�����ڴ�
	SPoint *queryTraGPU=NULL, **candidateTraGPU=NULL;
	int *candidateLengthGPU=NULL, *resultGPU=NULL;
	CUDA_CALL(hipMalloc((void**)&queryTraGPU, sizeof(SPoint)*queryLength));
	CUDA_CALL(hipMalloc((void**)&candidateLengthGPU, sizeof(int)*candidateNum));
	//CUDA_CALL(hipMalloc((void**)&resultGPU, sizeof(int)*candidateNum));

	SPoint **tempS = (SPoint**)malloc(sizeof(SPoint*)*candidateNum);
	for (int i = 0; i <= candidateNum - 1; i++) {
		CUDA_CALL(hipMalloc((void**)&tempS[i], sizeof(SPoint)*candidateLength[i]));
		
	}
	CUDA_CALL(hipMalloc((void***)&candidateTraGPU, sizeof(SPoint*)*candidateNum));
	CUDA_CALL(hipMemcpy(candidateTraGPU, tempS, candidateNum*sizeof(SPoint*), hipMemcpyHostToDevice));
	//
	time1.stop();
	std::cout << time1.elapse() << std::endl;
	time1.start();
	//
	//���ͨ���������ķ������ݹ켣�����Ҫ��켣�����洢
	//��GPU���ݹ켣��Ϣ
	CUDA_CALL(hipMemcpy(queryTraGPU, queryTra, queryLength*sizeof(SPoint), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(candidateLengthGPU, candidateLength, candidateNum*sizeof(int), hipMemcpyHostToDevice));
	
	for (int i = 0; i <= candidateNum - 1; i++) {
		CUDA_CALL(hipMemcpy(tempS[i], candidateTra[i], candidateLength[i] * sizeof(SPoint), hipMemcpyHostToDevice));
	}
	//for (int i = 0; i <= candidateNum - 1;i++)
	//	CUDA_CALL(hipMemcpy(candidateTraGPU[i], candidateTra[i], candidateLength[i]*sizeof(SPoint), hipMemcpyHostToDevice));
	CUDA_CALL(hipHostAlloc((void**)&result, candidateNum*sizeof(int), hipHostMallocWriteCombined | hipHostMallocMapped));
	CUDA_CALL(hipHostGetDevicePointer(&resultGPU, result, 0));
	time1.stop();
	std::cout << time1.elapse() << std::endl;
	time1.start();
	//ִ��kernel
	EDRDistance_1 <<<candidateNum, MAXTHREAD >>>(queryTraGPU, candidateTraGPU, candidateNum, queryLength, candidateLengthGPU, stateTableGPU, resultGPU);

	//ȡ���
	//result = (int*)malloc(candidateNum*sizeof(int));
	//CUDA_CALL(hipMemcpy(result, resultGPU, candidateNum*sizeof(int), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
//	for (int j = 0; j <= candidateNum - 1;j++)
//		std::cout << result[j] << std::endl;

	//free GPU!!!!!
	time1.stop();
	std::cout << time1.elapse() << std::endl;
	return 0;

}

