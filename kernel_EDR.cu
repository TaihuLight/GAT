#include "hip/hip_runtime.h"
//����EDR��GPU�����㷨
//zbw0046 3.22



#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include "ConstDefine.h"
#include "cudaKernel.h"
#include <assert.h>
#include <stdlib.h>
#include"hip/device_functions.h"
#include "WinTimer.h"

#define CUDA_CALL(x) { const hipError_t a = (x); if (a!= hipSuccess) { printf("\nCUDA Error: %s(err_num=%d)\n", hipGetErrorString(a), a); hipDeviceReset(); assert(0);}}


/*
���м���1�����ģdp
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
iter: �ڼ���dp��λ��outputIdx����������ȫ���ڴ�λ�ã�tra1��tra2�������켣����ǰ�����빲���ڴ棻
*/
//__global__ void DPforward(const int iter, const int* outputIdx,const SPoint *tra1,const SPoint *tra2) {
//	SPoint p1 = tra1[threadIdx.x];
//	SPoint p2 = tra2[iter - threadIdx.x - 1]; //�������ڴ��Ǿۼ����ʵ���
//	bool subcost;
//	if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
//		subcost = 0;
//	}
//	else
//		subcost = 1;
//
//}

/*
SPoint�汾
case1���켣����С��512
���м���n��DP
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
queryTra[],candidateTra[][]:�켣
stateTableGPU[][]:��ÿ��candidate��state��
result[]:����ÿ��candidate��EDR���
�Ż�����
1���켣����share memory����
2��ֱ�Ӵ��ݹ켣����ʹ��ָ��
*/
__global__ void EDRDistance_1(SPoint *queryTra, SPoint **candidateTra,int candidateNum,int queryLength,int *candidateLength,int** stateTableGPU,int *result) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	if (blockID >= candidateNum) return;
	if ((threadID >= candidateLength[blockID]) && (threadID >= queryLength)) return;
	const int lenT = candidateLength[blockID];
	//int iterNum = queryLength;
	//if (lenT > queryLength)
	//	iterNum = lenT;
	const int iterNum = queryLength + lenT - 1;
	__shared__ int state[2][MAXTHREAD]; //���ڴ洢ǰ���εĽ��
	state[0][0] = 0;
	state[1][0] = 1;
	state[1][1] = 1;
	//�������켣���򣬱�֤��һ���ȵڶ�����
	//���Ȱѹ켣���ڹ����ڴ���
	__shared__ SPoint queryTraS[MAXTHREAD];
	__shared__ SPoint traData[MAXTHREAD];
	if (threadID < lenT) {
		traData[threadID] = candidateTra[blockID][threadID];
	}
	if (threadID < queryLength) {
		queryTraS[threadID] = queryTra[threadID];
	}
	const SPoint *tra1, *tra2; //��֤tra1��tra2��
	int len1, len2;
	if (lenT >= queryLength) {
		tra1 = queryTraS;
		tra2 = traData;
		len1 = queryLength;
		len2 = lenT;
	}
	else
	{
		tra1 = traData;
		tra2 = queryTraS;
		len1 = lenT;
		len2 = queryLength;
	}

	int myState;
	for (int i = 0; i <= iterNum - 1; i++) {//��i��dp
		if (i < len1 - 1) {
			if (threadID <= i) {
				SPoint p1 = tra1[threadID];
				SPoint p2 = tra2[i - threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				//if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
				//	subcost = 0;
				//}
				//else
				//	subcost = 1;
				subcost = !(((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON);
				int state_ismatch = state[0][threadID] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID+1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_up;
				//ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
				//myState = (state_ismatch < state_up) * state_ismatch + (state_left < state_up) * state_up + (state_left >= state_up) * state_left;
				
			}
		}
		else if (i > iterNum - len1) {
			if (threadID <= iterNum - i - 1) {
				SPoint p1 = tra1[threadID+len1-(iterNum-i)];
				SPoint p2 = tra2[len2-1-threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
					subcost = 0;
				}
				else
					subcost = 1;
				int state_ismatch = state[0][threadID+1] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID + 1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_up;
			}
		}
		else
		{
			if (threadID < len1) {
				SPoint p1 = tra1[threadID];
				SPoint p2 = tra2[i-threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
					subcost = 0;
				}
				else
					subcost = 1;
				int state_ismatch = state[0][threadID] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID + 1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_up;
			}
		}
		//дmyState��share�ڴ�,ckecked
		int startidx;
		//���Ƚ�������д��ȫ���ڴ棬ȫд
		//startidx�Ǿɵ�����Ӧ����ȫ���ڴ��е�ַ����i-2����
		//����Ӧд��ȫ���ڴ����ʼλ��

		if (i-2 < len1 - 2) {
			startidx = (i-2 + 2)*(i-2 + 3) / 2;
			if (threadID <= i) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}
		else if (i-2 >= iterNum - len1) {
			startidx = (len1 + 1)*(len2 + 1) - (iterNum - (i-2))*(iterNum - (i-2) + 1) / 2;
			if (threadID <= iterNum - i + 1 ) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}
		else
		{
			startidx = (len1 + 1)*((i - 2) - (len1 - 2)) + len1*(len1 + 1) / 2;
			if (threadID <= len1) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}

		//�ƶ������ݵ�������
		state[0][threadID] = state[1][threadID];
		//д��������
		if (i < len1-1) {
			if (threadID <= i)
				state[1][threadID + 1] = myState;
			if (threadID == 0) {
				state[1][0] = i + 2;
				state[1][i + 2] = i + 2;
			}
		}
		else if (i >= iterNum - len1) {
			if (threadID <= iterNum - i - 1)
				state[1][threadID] = myState;
		}
		else
		{
			if (threadID < len1)
				state[1][threadID + 1] = myState;
			if (threadID == 0) {
				state[1][0] = i + 2;
			}
		}
		__syncthreads();
	}
	//�����������һ�μ���һ�����ɽ���0��ɵ�
	if (threadID == 0)
		result[blockID] = myState;
}


//__global__ void testSharedMemory()
//{
//	__shared__ SPoint queryTraS[MAXLENGTH];
//	__shared__ SPoint traData[MAXLENGTH];
//	__shared__ SPoint traData2[MAXLENGTH];
//	SPoint s;
//	s.x = 4;
//	s.y = 5;
//	traData[1535] = s;
//	queryTraS[1535] = s;
//	traData2[1535] = s;
//}


/*
SPoint�汾
ͬʱ�������ɸ�query��EDR��������һ��EDR����Ϊ��λ��ÿ��block����һ��EDR��thread����һ��б����state�Ĳ��м��㡣
case1���켣���ȿɳ���512������ѭ���������512�ġ�
���м���n��DP
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
queryTaskNum:�ܹ��м���EDR��������
queryTaskInfo[]��ÿ��task��Ӧ��qID��candidateID��Ϣ����struct�洢
queryTra[],candidateTra[]:�켣���ݣ�candidateTra��֤���ڲ��켣���ظ�
queryTraOffset[],candidateTraOffset[]:ÿ���켣��offset��candidateTra��֤���ڲ��켣���ظ�
queryLength[],candidateLength[]:ÿ���켣�ĳ��ȣ���ʵoffset������ǳ��ȣ�����idx������Ķ�Ӧ
����candidateLength[id]�ǵ�id��candidate Traj�ĳ���
stateTableGPU[][]:��ÿ��candidate��state��
result[]:����ÿ��candidate��EDR���
�Ż�����
1���켣����share memory����
2��ֱ�Ӵ��ݹ켣����ʹ��ָ��
*/



__global__ void EDRDistance_Batch(int queryTaskNum, TaskInfoTableForSimilarity* taskInfoTable, SPoint *queryTra, int* queryTraOffset, SPoint** candidateTraOffsets, int* queryLength, int *candidateLength, int *result) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	if (blockID >= queryTaskNum) return;
	int thisQueryID = taskInfoTable[blockID].qID;
	int thisQueryLength = queryLength[thisQueryID];
	if ((threadID >= candidateLength[blockID]) && (threadID >= thisQueryLength)) return;
	const int lenT = candidateLength[blockID];
	//int iterNum = queryLength;
	//if (lenT > queryLength)
	//	iterNum = lenT;
	const int iterNum = thisQueryLength + lenT - 1;
	__shared__ int state[2][MAXLENGTH+1]; //���ڴ洢ǰ���εĽ����ռ��8KB��
	state[0][0] = 0;
	state[1][0] = 1;
	state[1][1] = 1;
	//�������켣���򣬱�֤��һ���ȵڶ�����
	//���Ȱѹ켣���ڹ����ڴ���
	//����������share memory�Ƿ��õ����⣬����д����64KB��Ȼ��K80�ƺ���512KB
	//�����64KB�Ļ���ÿ���켣�1024���㣨�����켣��ռ��24KB��
	//__shared__ SPoint queryTraS[MAXLENGTH];
	//__shared__ SPoint traData[MAXLENGTH];


	//for (int i = 0; i <= lenT - 1;i+=MAXTHREAD)
	//{
	//	if(threadID+i<lenT)
	//	{
	//		traData[threadID + i] = SPoint(candidateTraOffsets[blockID][threadID + i]);
	//	}
	//}

	SPoint* queryTraBaseAddr = queryTra + queryTraOffset[thisQueryID];
	//for (int i = 0; i <= thisQueryLength - 1;i+=MAXTHREAD)
	//{
	//	if(threadID+i<thisQueryLength)
	//	{
	//		queryTraS[threadID + i] = *(queryTraBaseAddr + threadID + i);
	//	}
	//}
	SPoint *queryTraS = queryTraBaseAddr;
	SPoint *traData = candidateTraOffsets[blockID];
	const SPoint *tra1, *tra2; //��֤tra1��tra2��
	int len1, len2;
	if (lenT >= thisQueryLength) {
		tra1 = queryTraS;
		tra2 = traData;
		len1 = thisQueryLength;
		len2 = lenT;
	}
	else
	{
		tra1 = traData;
		tra2 = queryTraS;
		len1 = lenT;
		len2 = thisQueryLength;
	}

	int myState[5];
	int nodeID;
	for (int i = 0; i <= iterNum - 1; i++) {//��i��dp
		if (i < len1 - 1) {
			for (int startIdx = 0; startIdx <= i; startIdx += MAXTHREAD) {
				nodeID = startIdx + threadID;
				if (nodeID <= i) {
					SPoint p1 = tra1[nodeID];
					SPoint p2 = tra2[i - nodeID]; //�������ڴ��Ǿۼ����ʵ���
					bool subcost;
					if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
						subcost = 0;
					}
					else
						subcost = 1;
					//subcost = !(((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON);
					int state_ismatch = state[0][nodeID] + subcost;
					int state_up = state[1][nodeID] + 1;
					int state_left = state[1][nodeID + 1] + 1;
					bool c1 = ((state_ismatch < state_up) && (state_ismatch < state_left));
					bool c2 = ((state_left < state_up) && ((state_left < state_ismatch)));
					//ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
					myState[nodeID / MAXTHREAD] = c1 * state_ismatch + c2 * state_left + !(c1 || c2) * state_up;
					//if ((state_ismatch < state_up) && (state_ismatch < state_left))
					//	myState[nodeID/MAXTHREAD] = state_ismatch;
					//else if ((state_left < state_up) && ((state_left < state_ismatch)))
					//	myState[nodeID / MAXTHREAD] = state_left;
					//else
					//	myState[nodeID / MAXTHREAD] = state_up;
					////ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
					//myState[nodeID / MAXTHREAD] = (state_ismatch < state_up) && (state_ismatch < state_left) * state_ismatch + ((state_left < state_up) && ((state_left < state_ismatch))) * state_left + !(((state_ismatch < state_up) && (state_ismatch < state_left))||(((state_left < state_up) && ((state_left < state_ismatch))))) * state_up;
				}
			}
		}
		else if (i > iterNum - len1) {
			for (int startIdx = 0; startIdx <= iterNum - i - 1; startIdx += MAXTHREAD) {
				nodeID = startIdx + threadID;
				if (nodeID <= iterNum - i - 1) {
					SPoint p1 = tra1[nodeID + len1 - (iterNum - i)];
					SPoint p2 = tra2[len2 - 1 - nodeID]; //�������ڴ��Ǿۼ����ʵ���
					bool subcost;
					if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
						subcost = 0;
					}
					else
						subcost = 1;
					int state_ismatch = state[0][nodeID + 1] + subcost;
					int state_up = state[1][nodeID] + 1;
					int state_left = state[1][nodeID + 1] + 1;
					//if (state_ismatch < state_up)
					//	myState[nodeID / MAXTHREAD] = state_ismatch;
					//else if (state_left < state_up)
					//	myState[nodeID / MAXTHREAD] = state_left;
					//else
					//	myState[nodeID / MAXTHREAD] = state_up;
					bool c1 = ((state_ismatch < state_up) && (state_ismatch < state_left));
					bool c2 = ((state_left < state_up) && ((state_left < state_ismatch)));
					//ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
					myState[nodeID / MAXTHREAD] = c1 * state_ismatch + c2 * state_left + !(c1 || c2) * state_up;
				}
			}
		}
		else
		{
			for (int startIdx = 0; startIdx <= len1; startIdx += MAXTHREAD) {
				nodeID = startIdx + threadID;
				if (nodeID <= len1) {
					SPoint p1 = tra1[nodeID];
					SPoint p2 = tra2[i - nodeID]; //�������ڴ��Ǿۼ����ʵ���
					bool subcost;
					if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
						subcost = 0;
					}
					else
						subcost = 1;
					int state_ismatch = state[0][nodeID] + subcost;
					int state_up = state[1][nodeID] + 1;
					int state_left = state[1][nodeID + 1] + 1;
					//if (state_ismatch < state_up)
					//	myState[nodeID / MAXTHREAD] = state_ismatch;
					//else if (state_left < state_up)
					//	myState[nodeID / MAXTHREAD] = state_left;
					//else
					//	myState[nodeID / MAXTHREAD] = state_up;
					bool c1 = ((state_ismatch < state_up) && (state_ismatch < state_left));
					bool c2 = ((state_left < state_up) && ((state_left < state_ismatch)));
					//ȥ��if�ı�﷽ʽ���Ƿ�����������ܣ�
					myState[nodeID / MAXTHREAD] = c1 * state_ismatch + c2 * state_left + !(c1 || c2) * state_up;
				}
			}
		}
		//дmyState��share�ڴ�,ckecked
		
		//int startidx;
		////���Ƚ�������д��ȫ���ڴ棬ȫд
		////startidx�Ǿɵ�����Ӧ����ȫ���ڴ��е�ַ����i-2����
		////����Ӧд��ȫ���ڴ����ʼλ��
		//// 7.2 ���֣��ƺ�stateTableGPU������д������������
		////////

		//if (i - 2 < len1 - 2) {
		//	startidx = (i - 2 + 2)*(i - 2 + 3) / 2;
		//	for (int Idx = 0; Idx <= i; Idx += MAXTHREAD) {
		//		//if (threadID <= i) {
		//		if(Idx + threadID <= i){
		//			//stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
		//			stateTableGPU[blockID][Idx + threadID + startidx] = state[0][threadID + Idx];
		//		}
		//	}
		//}
		//else if (i - 2 >= iterNum - len1) {
		//	startidx = (len1 + 1)*(len2 + 1) - (iterNum - (i - 2))*(iterNum - (i - 2) + 1) / 2;
		//	for (int Idx = 0; Idx <= iterNum - i + 1; Idx += MAXTHREAD) {
		//		//if (threadID <= iterNum - i + 1) {
		//		if (threadID + Idx <= iterNum - i + 1) {
		//			//stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
		//			stateTableGPU[blockID][Idx + threadID + startidx] = state[0][Idx + threadID];
		//		}
		//	}
		//}
		//else
		//{
		//	startidx = (len1 + 1)*((i - 2) - (len1 - 2)) + len1*(len1 + 1) / 2;
		//	for (int Idx = 0; Idx <= len1; Idx += MAXTHREAD) {
		//		//if (threadID <= len1) {
		//		if (threadID + Idx <= len1) {
		//			// stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
		//			stateTableGPU[blockID][Idx + threadID + startidx] = state[0][Idx + threadID];
		//		}
		//	}
		//}

		//�ƶ������ݵ�������
		for (int Idx = 0; Idx < MAXLENGTH;Idx+=MAXTHREAD)
		{
			state[0][threadID+Idx] = state[1][threadID+Idx];
		}
		//state[0][threadID] = state[1][threadID];

		//д��������
		if (i < len1 - 1) {
			//if (threadID <= i)
			//	state[1][threadID + 1] = myState;
			//if (threadID == 0) {
			//	state[1][0] = i + 2;
			//	state[1][i + 2] = i + 2;
			//}
			for (int Idx = 0; Idx <= i; Idx += MAXTHREAD) {
				if (threadID + Idx <= i)
					state[1][Idx + threadID + 1] = myState[Idx/MAXTHREAD];
			}
			if (threadID == 0) {
				state[1][0] = i + 2;
				state[1][i + 2] = i + 2;
			}
		}
		else if (i >= iterNum - len1) {
			//if (threadID <= iterNum - i - 1)
			//	state[1][threadID] = myState;
			for (int Idx = 0; Idx <= iterNum - i - 1; Idx += MAXTHREAD) {
				if (threadID + Idx <= iterNum - i - 1)
					state[1][threadID + Idx] = myState[Idx / MAXTHREAD];
			}
		}
		else
		{
			//if (threadID < len1)
			//	state[1][threadID + 1] = myState;
			//if (threadID == 0) {
			//	state[1][0] = i + 2;
			//}
			for (int Idx = 0; Idx <= len1; Idx += MAXTHREAD) {
				if (threadID + Idx < len1)
					state[1][Idx + threadID + 1] = myState[Idx / MAXTHREAD];
			}
			if (threadID == 0) {
				state[1][0] = i + 2;
			}
		}
		__syncthreads();
	}
	//�����������һ�μ���һ�����ɽ���0��ɵ�
	if (threadID == 0 && blockID < queryTaskNum)
		result[blockID] = myState[0];
}

int EDRDistance_Batch_Handler(int queryTaskNum, TaskInfoTableForSimilarity* taskInfoTable, SPoint *queryTra, int* queryTraOffset, SPoint** candidateTraOffsets, int* queryLength, int *candidateLength, int *result, hipStream_t *stream)
{
	EDRDistance_Batch <<<queryTaskNum, MAXTHREAD,0 , *stream >>>(queryTaskNum, taskInfoTable, queryTra, queryTraOffset, candidateTraOffsets, queryLength, candidateLength, result);
	return 0;
}

__device__ inline int binary_search_intPair(intPair* temp, int left,int right,int val)
{
	int mid = (left + right) / 2;
	while(left<=right)
	{
		mid = (left + right) / 2;
		if (temp[mid].int_1 == val)
			return temp[mid].int_2;
		else if (temp[mid].int_1 > val)
		{
			right = mid-1;
		}
		else
			left = mid+1;
	}
	return 0;
}

__device__ inline int binary_search_intPair_Neighbor(intPair* temp, int left, int right, int val)
{
	int mid = (left + right) / 2;
	while (left <= right)
	{
		mid = (left + right) / 2;
		if (temp[mid].int_1 == val)
			return mid;
		else if (temp[mid].int_1 > val)
		{
			right = mid - 1;
		}
		else
			left = mid + 1;
	}
	return -1;
}

// -1Ϊû�ҵ�
__device__ inline int binary_search_int(int* temp, int left, int right, int val)
{
	int mid = (left + right) / 2;
	while (left <= right)
	{
		mid = (left + right) / 2;
		if (temp[mid] == val)
			return mid;
		else if (temp[mid] > val)
		{
			right = mid - 1;
		}
		else
			left = mid + 1;
	}
	return -1;
}

__device__ inline int getIdxFromXYGPU(int x, int y)
{
	int lenx, leny;
	if (x == 0)
		lenx = 1;
	else
	{
		lenx = int(log2f(x)) + 1;
	}
	if (y == 0)
		leny = 1;
	else
		leny = int(log2f(y)) + 1;
	int result = 0;
	int xbit = 1, ybit = 1;
	for (int i = 1; i <= 2 * max(lenx, leny); i++)
	{
		if ((i & 1) == 1) //����
		{
			result += (x >> (xbit - 1) & 1) * (1 << (i - 1));
			xbit = xbit + 1;
		}
		else //ż��
		{
			result += (y >> (ybit - 1) & 1) * (1 << (i - 1));
			ybit = ybit + 1;
		}
	}
	return result;
}

__device__ inline int findNeighborGPU(int cellNum, int cellID, int * neighborID)
{
	int x = 0, y = 0;
	for (int bit = 0; bit <= int(log2f(cellNum)) - 1; bit++) {
		if (bit % 2 == 0) {
			//����λ
			x += ((cellID >> bit)&(1))*(1 << (bit / 2));
		}
		else {
			//ż��λ
			y += ((cellID >> bit)&(1))*(1 << (bit / 2));
		}
	}
	int cnt = 0;
	for (int xx = x - 1; xx <= x + 1; xx++) {
		for (int yy = y - 1; yy <= y + 1; yy++) {
			if ((xx != x) || (yy != y))
				neighborID[cnt++] = getIdxFromXYGPU(xx, yy);
			//printf("%d\t", cnt);
		}
	}
	return 0;
}

__device__ inline bool isPositive(short x)
{
	return x >= 0;
}

__global__ void Calculate_FD_Sparse(intPair* queryFVGPU, intPair* FVinfo, intPair* FVTable, intPair* SubbedArray, intPair* SubbedArrayOffset, int SubbedArrayJump, int queryCellLength, int startTrajIdx, int checkNum, int cellNum, int trajNumInDB, int nonZeroFVNumInDB, short* FDistance)
{
	//��һ�׶Σ����м���
	const int MAX_QUERY_CELLNUMBER = 512;
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int threadIDGlobal = blockDim.x*blockID + threadID;

	__shared__ intPair queryCellTraj[MAX_QUERY_CELLNUMBER];
	__shared__ intPair dbCellTraj[MAX_QUERY_CELLNUMBER];
	//cellchecked��¼��query�г��ֵ�cell��ţ������ڷ��������ʱ�����ǲ����Ѿ������ˡ��Ժ�����ڹ鲢���и��ô˱�����
	__shared__ int cellChecked[MAX_QUERY_CELLNUMBER];
	for (int i = 0; i <= queryCellLength - 1; i += MAXTHREAD) {
		if (threadID+i < queryCellLength)
		{
			queryCellTraj[threadID + i] = queryFVGPU[threadID + i];
		}
	}
	int dbTrajStartIdx = FVinfo[startTrajIdx + blockID].int_2;
	int dbTrajEndIdx;
	if (blockID + startTrajIdx == trajNumInDB - 1)
		dbTrajEndIdx = nonZeroFVNumInDB - 1;
	else
		dbTrajEndIdx = FVinfo[startTrajIdx + blockID + 1].int_2 - 1;
	
	for (int i = 0; i <= dbTrajEndIdx - dbTrajStartIdx;i+=MAXTHREAD)
	{
		if (threadID + i <= dbTrajEndIdx - dbTrajStartIdx)
			dbCellTraj[threadID + i] = FVTable[dbTrajStartIdx + threadID + i];
	}
	//1.1:��query��ȥdb
	for (int i = 0; i < queryCellLength; i += MAXTHREAD)
	{
		if (threadID + i < queryCellLength) {
			int find = binary_search_intPair(dbCellTraj, 0, dbTrajEndIdx - dbTrajStartIdx, queryCellTraj[threadID + i].int_1);
			cellChecked[threadID + i] = queryCellTraj[threadID + i].int_1;
			SubbedArray[SubbedArrayJump * blockID + threadID + i].int_1 = queryCellTraj[threadID + i].int_1;
			SubbedArray[SubbedArrayJump * blockID + threadID + i].int_2 = queryCellTraj[threadID + i].int_2 - find;
		}
		if (threadID == 0) {
			SubbedArrayOffset[blockID].int_1 = queryCellLength - 1;
			SubbedArrayOffset[blockID].int_2 = queryCellLength + dbTrajEndIdx - dbTrajStartIdx;
		}
	}
	//1.2����db��ȥquery��ע��Ӹ���
	for (int i = 0; i <= dbTrajEndIdx - dbTrajStartIdx;i+=MAXTHREAD)
	{
		if(threadID + i <= dbTrajEndIdx - dbTrajStartIdx)
		{
			intPair cellNo = dbCellTraj[threadID + i];
			int find = binary_search_int(cellChecked, 0, queryCellLength - 1, cellNo.int_1);
			if (find == -1)
			{
				SubbedArray[SubbedArrayJump * blockID + queryCellLength + threadID + i].int_1 = cellNo.int_1;
				SubbedArray[SubbedArrayJump * blockID + queryCellLength + threadID + i].int_2 = -cellNo.int_2;
			}
			else
				SubbedArray[SubbedArrayJump * blockID + queryCellLength + threadID + i].int_1 = -1;
		}
	}
	__syncthreads();
	//�ڶ��׶Σ��������ڣ�������
	//����׶θ�Ϊÿ��thread����һ��FD
	//2.1���ϲ�ÿ��subbedArray
	if (threadIDGlobal < checkNum) {
		int startMergeIdx = SubbedArrayOffset[threadIDGlobal].int_1 + 1;
		int endMergeIdx = SubbedArrayOffset[threadIDGlobal].int_2;
		int frontPtr = startMergeIdx;
		for (int i = startMergeIdx; i <= endMergeIdx;i++)
		{
			if(SubbedArray[SubbedArrayJump * threadIDGlobal + i].int_1 != -1)
			{
				SubbedArray[SubbedArrayJump * threadIDGlobal + frontPtr] = SubbedArray[SubbedArrayJump * threadIDGlobal + i];
				frontPtr++;
			}
		}
		SubbedArrayOffset[threadIDGlobal].int_2 = frontPtr-1;
	}
	//2.2 ��������
	int neighborsID[8];
	//cell����ָ�ڼ���Ԫ��
	for (int cell = 0; cell <= SubbedArrayOffset[threadIDGlobal].int_2; cell++)
	{
		findNeighborGPU(cellNum, cell, neighborsID);
		//for (int i = 0; i <= 7; i++)
		//	neighborsID[i] = 11;
		for (int i = 0; i <= 7; i++)
		{
			int find = binary_search_intPair_Neighbor(&SubbedArray[SubbedArrayJump * threadIDGlobal], 0, SubbedArrayOffset[threadIDGlobal].int_1, neighborsID[i]);
			if(find == -1){
				find = binary_search_intPair_Neighbor(&SubbedArray[SubbedArrayJump * threadIDGlobal], SubbedArrayOffset[threadIDGlobal].int_1 + 1, SubbedArrayOffset[threadIDGlobal].int_2, neighborsID[i]);
			}
			// �����-1��˵�����neighbor��0�����ô���
			if(find != -1)
			{
				if (isPositive(SubbedArray[SubbedArrayJump * threadIDGlobal + cell].int_2) != isPositive(SubbedArray[SubbedArrayJump * threadIDGlobal + find].int_2))
				{
					if (fabsf(SubbedArray[SubbedArrayJump * threadIDGlobal + cell].int_2) > fabsf(SubbedArray[SubbedArrayJump * threadIDGlobal + find].int_2))
					{
						SubbedArray[SubbedArrayJump * threadIDGlobal + cell].int_2 = SubbedArray[SubbedArrayJump * threadIDGlobal + cell].int_2 + SubbedArray[SubbedArrayJump * threadIDGlobal + find].int_2;
						SubbedArray[SubbedArrayJump * threadIDGlobal + find].int_2 = 0;
					}
					else {
						SubbedArray[SubbedArrayJump * threadIDGlobal + find].int_2 = SubbedArray[SubbedArrayJump * threadIDGlobal + find].int_2 + SubbedArray[SubbedArrayJump * threadIDGlobal + cell].int_2;
						SubbedArray[SubbedArrayJump * threadIDGlobal + cell].int_2 = 0;
						break;
					}
				}
			}
		}
	}
	__syncthreads();
	//�����׶Σ�ͳ����������
	//��Ȼ��ÿ��block����һ��FD�ļ���
	if (blockID >= checkNum)
		return;
	int *tempsumPosi = (int*)queryCellTraj;
	int *tempsumNega = (int*)dbCellTraj;
	tempsumPosi[threadID] = 0;
	tempsumNega[threadID] = 0;
	for (int i = 0; i <= SubbedArrayOffset[blockID].int_2; i += MAXTHREAD)
	{
		if(i+threadID <= SubbedArrayOffset[blockID].int_2)
		{
			tempsumPosi[threadID] += (isPositive(SubbedArray[SubbedArrayJump * blockID + i + threadID].int_2)*SubbedArray[SubbedArrayJump * blockID + i + threadID].int_2);
			tempsumNega[threadID] += (-(!isPositive(SubbedArray[SubbedArrayJump * blockID + i + threadID].int_2))*SubbedArray[SubbedArrayJump * blockID + i + threadID].int_2);
		}
	}
	__shared__ int sizeOfTempSum;
	if (threadID == 0)
		sizeOfTempSum = MAXTHREAD;
	__syncthreads();
	while ((sizeOfTempSum>1))
	{
		if (threadID <= (sizeOfTempSum >> 1) - 1)
		{
			tempsumPosi[threadID] = tempsumPosi[threadID] + tempsumPosi[threadID + (sizeOfTempSum >> 1)];
			tempsumNega[threadID] = tempsumNega[threadID] + tempsumNega[threadID + (sizeOfTempSum >> 1)];
		}
		__syncthreads();
		if (threadID == 0)
			sizeOfTempSum = (sizeOfTempSum >> 1);
		__syncthreads();
	}
	if (threadID == 0)
		FDistance[blockID] = (tempsumPosi[0] > tempsumNega[0]) ? tempsumPosi[0] : tempsumNega[0];
}

//ÿ��block����һ��FD�ļ���
__global__ void Calculate_FD_NonColumn(short* queryFVGPU, intPair* FVinfo, intPair* FVTable, int startTrajIdx, int checkNum,int cellNum, int trajNumInDB, int nonZeroFVNumInDB, short* FDistance)
{
	//��һ�׶Σ����м���
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int threadIDGlobal = blockDim.x*blockID + threadID;
	if (blockID >= checkNum)
		return;
	__shared__ intPair taskInfo;
	if(threadID == 0)
		taskInfo = FVinfo[blockID + startTrajIdx];
	int nextCnt;
	if (blockID + startTrajIdx == trajNumInDB - 1)
		nextCnt = nonZeroFVNumInDB;
	else
		nextCnt = FVinfo[blockID + startTrajIdx + 1].int_2;
	__syncthreads();
	for (int i = 0; i <= (cellNum-1);i+=MAXTHREAD)
	{
		int find = binary_search_intPair(FVTable, taskInfo.int_2, (nextCnt - 1), (i + threadID));
		//int find = 1;
		//int k = cellNum*blockID + (i + threadID);
		//queryFVGPU[cellNum*blockID + (i + threadID)] = 2;
		queryFVGPU[cellNum*blockID + (i + threadID)] = queryFVGPU[cellNum*blockID + (i + threadID)] - find;
	}
	//�ڶ��׶Σ��������ڣ�������
	//����׶θ�Ϊÿ��thread����һ��FD
	int neighborsID[8];
	for (int cell = 0; cell <= cellNum - 1;cell++)
	{
		//ֻ��Ҫһ�����߳̾�����
		if (threadIDGlobal >= checkNum)
			break;
		if (queryFVGPU[cellNum*threadIDGlobal + cell] != 0)
		{
			findNeighborGPU(cellNum, cell, neighborsID);
			//for (int i = 0; i <= 7; i++)
			//	neighborsID[i] = 11;
			for (int i = 0; i <= 7; i++)
			{
				if (isPositive(queryFVGPU[cellNum*threadIDGlobal + cell]) != isPositive(queryFVGPU[cellNum*threadIDGlobal + neighborsID[i]])){
					if (fabsf(queryFVGPU[cellNum*threadIDGlobal + cell]) > fabsf(queryFVGPU[cellNum*threadIDGlobal + neighborsID[i]]))
					{
						queryFVGPU[cellNum*threadIDGlobal + cell] = queryFVGPU[cellNum*threadIDGlobal + cell] + queryFVGPU[cellNum*threadIDGlobal + neighborsID[i]];
						queryFVGPU[cellNum*threadIDGlobal + neighborsID[i]] = 0;
					}
					else
					{
						queryFVGPU[cellNum*threadIDGlobal + neighborsID[i]] = queryFVGPU[cellNum*threadIDGlobal + neighborsID[i]] + queryFVGPU[cellNum*threadIDGlobal + cell];
						queryFVGPU[cellNum*threadIDGlobal + cell] = 0;
						break;
					}
				}
			}
		}
	}
	__syncthreads();
	//�����׶Σ�ͳ����������
	//��Ȼ��ÿ��block����һ��FD�ļ���
	__shared__ int tempsumPosi[MAXTHREAD], tempsumNega[MAXTHREAD];
	tempsumPosi[threadID] = 0;
	tempsumNega[threadID] = 0;
	for (int i = 0; i <= cellNum - 1;i+=MAXTHREAD)
	{
		tempsumPosi[threadID] += (isPositive(queryFVGPU[blockID*cellNum + (i + threadID)])*queryFVGPU[blockID*cellNum + (i + threadID)]);
		tempsumNega[threadID] += (-(!isPositive(queryFVGPU[blockID*cellNum + (i + threadID)]))*queryFVGPU[blockID*cellNum + (i + threadID)]);
	}
	__shared__ int sizeOfTempSum;
	if(threadID==0)
		sizeOfTempSum = MAXTHREAD;
	__syncthreads();
	while((sizeOfTempSum>1))
	{
		if (threadID <= (sizeOfTempSum >> 1)-1)
		{
			tempsumPosi[threadID] = tempsumPosi[threadID] + tempsumPosi[threadID + (sizeOfTempSum>>1)];
			tempsumNega[threadID] = tempsumNega[threadID] + tempsumNega[threadID + (sizeOfTempSum>>1)];
		}
		__syncthreads();
		if(threadID == 0)
			sizeOfTempSum = (sizeOfTempSum >> 1);
		__syncthreads();
	}
	if (threadID == 0)
		FDistance[blockID] = (tempsumPosi[0] > tempsumNega[0]) ? tempsumPosi[0] : tempsumNega[0];

}

//SubbedArrayJump��SubbedArray��ÿһ���ж��ٸ�Ԫ�أ�������idx��
int Similarity_Pruning_Handler(intPair* queryFVGPU, intPair* FVinfo, intPair* FVTable, intPair* SubbedArray, intPair* SubbedArrayOffset,int SubbedArrayJump, int queryCellLength, int startTrajIdx, int checkNum, int cellNum, int trajNumInDB, int nonZeroFVNumInDB, short* FDistance, hipStream_t stream)
{
#ifdef NOT_COLUMN_ORIENTED
	Calculate_FD_NonColumn <<<checkNum, MAXTHREAD, 0, stream >>>(queryFVGPU, FVinfo, FVTable, startTrajIdx, checkNum, cellNum, trajNumInDB, nonZeroFVNumInDB, FDistance);
#else
	Calculate_FD_Sparse <<<checkNum, MAXTHREAD, 0, stream >>>(queryFVGPU, FVinfo, FVTable, SubbedArray, SubbedArrayOffset, SubbedArrayJump, queryCellLength, startTrajIdx, checkNum, cellNum, trajNumInDB, nonZeroFVNumInDB, FDistance);
#endif
	return 0;
}


/*
//�Ȱ����ܷ���һ��SMִ��һ��DP�����������ٷֱ��������kernel
//constructing...
���Ż���
1��queryTra��queryLength����candidateLength����ͨ����ֵ�ķ�ʽֱ�Ӵ��ݵ�SM�ļĴ���������ȫ���ڴ��ʹ��

*/
int handleEDRdistance(SPoint *queryTra, SPoint **candidateTra, int candidateNum, int queryLength, int *candidateLength,int *result) {
	MyTimer time1;
	time1.start();

	int** stateTableGPU=NULL;
	//��GPU��Ϊ״̬������ڴ�
	int** temp=NULL;
	temp = (int**)malloc(sizeof(int*)*candidateNum);
	for (int i = 0; i <= candidateNum - 1; i++) {
		CUDA_CALL(hipMalloc((void**)&temp[i], sizeof(int)*(candidateLength[i] + 1)*(queryLength + 1)));
	}
	CUDA_CALL(hipMalloc((void***)&stateTableGPU, sizeof(int*)*candidateNum));
	CUDA_CALL(hipMemcpy(stateTableGPU, temp, candidateNum*sizeof(int*), hipMemcpyHostToDevice));

	//Ϊ�洢�Ĺ켣��Ϣ�����ڴ�
	SPoint *queryTraGPU=NULL, **candidateTraGPU=NULL;
	int *candidateLengthGPU=NULL, *resultGPU=NULL;
	CUDA_CALL(hipMalloc((void**)&queryTraGPU, sizeof(SPoint)*queryLength));
	CUDA_CALL(hipMalloc((void**)&candidateLengthGPU, sizeof(int)*candidateNum));
	//CUDA_CALL(hipMalloc((void**)&resultGPU, sizeof(int)*candidateNum));

	SPoint **tempS = (SPoint**)malloc(sizeof(SPoint*)*candidateNum);
	for (int i = 0; i <= candidateNum - 1; i++) {
		CUDA_CALL(hipMalloc((void**)&tempS[i], sizeof(SPoint)*candidateLength[i]));
		
	}
	CUDA_CALL(hipMalloc((void***)&candidateTraGPU, sizeof(SPoint*)*candidateNum));
	CUDA_CALL(hipMemcpy(candidateTraGPU, tempS, candidateNum*sizeof(SPoint*), hipMemcpyHostToDevice));
	//
	time1.stop();
	std::cout << time1.elapse() << std::endl;
	time1.start();
	//
	//���ͨ���������ķ������ݹ켣�����Ҫ��켣�����洢
	//��GPU���ݹ켣��Ϣ
	CUDA_CALL(hipMemcpy(queryTraGPU, queryTra, queryLength*sizeof(SPoint), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(candidateLengthGPU, candidateLength, candidateNum*sizeof(int), hipMemcpyHostToDevice));
	
	for (int i = 0; i <= candidateNum - 1; i++) {
		CUDA_CALL(hipMemcpy(tempS[i], candidateTra[i], candidateLength[i] * sizeof(SPoint), hipMemcpyHostToDevice));
	}
	//for (int i = 0; i <= candidateNum - 1;i++)
	//	CUDA_CALL(hipMemcpy(candidateTraGPU[i], candidateTra[i], candidateLength[i]*sizeof(SPoint), hipMemcpyHostToDevice));
	CUDA_CALL(hipHostAlloc((void**)&result, candidateNum*sizeof(int), hipHostMallocWriteCombined | hipHostMallocMapped));
	CUDA_CALL(hipHostGetDevicePointer(&resultGPU, result, 0));
	time1.stop();
	std::cout << time1.elapse() << std::endl;
	time1.start();
	//ִ��kernel
	EDRDistance_1 <<<candidateNum, MAXTHREAD >>>(queryTraGPU, candidateTraGPU, candidateNum, queryLength, candidateLengthGPU, stateTableGPU, resultGPU);

	//ȡ���
	//result = (int*)malloc(candidateNum*sizeof(int));
	//CUDA_CALL(hipMemcpy(result, resultGPU, candidateNum*sizeof(int), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
//	for (int j = 0; j <= candidateNum - 1;j++)
//		std::cout << result[j] << std::endl;

	//free GPU!!!!!
	time1.stop();
	std::cout << time1.elapse() << std::endl;
	return 0;

}

