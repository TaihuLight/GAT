#include "hip/hip_runtime.h"
//����EDR��GPU�����㷨
//zbw0046 3.22



#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include "ConstDefine.h"

#define CUDA_CALL(x) { const hipError_t a = (x); if (a!= hipSuccess) { printf("\nCUDA Error: %s(err_num=%d)\n", hipGetErrorString(a), a); hipDeviceReset(); assert(0);}}
#define EPSILON 10
#define MAXTHREAD 512

typedef struct SPoint {
	float x;
	float y;
	uint32_t tID;
}SPoint;

/*
���м���1�����ģdp
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
iter: �ڼ���dp��λ��outputIdx����������ȫ���ڴ�λ�ã�tra1��tra2�������켣����ǰ�����빲���ڴ棻
*/
__global__ void DPforward(const int iter, const int* outputIdx,const SPoint *tra1,const SPoint *tra2) {
	SPoint p1 = tra1[threadIdx.x];
	SPoint p2 = tra2[iter - threadIdx.x - 1]; //�������ڴ��Ǿۼ����ʵ���
	bool subcost;
	if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
		subcost = 0;
	}
	else
		subcost = 1;

}

/*
case1���켣����С��512
���м���n��DP
��Ҫ��ǰ����ǰ����dp�Ľ���������ڹ����ڴ���
queryTra[],candidateTra[][]:�켣
stateTableGPU[][]:��ÿ��candidate��state��
result[]:����ÿ��candidate��EDR���
�Ż�����
1���켣����share memory����
*/
__global__ void EDRDistance_1(const SPoint *queryTra, const SPoint **candidateTra,const int candidateNum,const int queryLength,const int *candidateLength,int** stateTableGPU,int *result) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	if (blockID >= candidateNum) return;
	if ((threadID >= candidateLength[blockID]) && (threadID >= queryLength)) return;
	const int lenT = candidateLength[blockID];
	//int iterNum = queryLength;
	//if (lenT > queryLength)
	//	iterNum = lenT;
	const int iterNum = queryLength + lenT - 1;
	__shared__ int state[2][MAXTHREAD]; //���ڴ洢ǰ���εĽ��
	state[0][0] = 0;
	state[1][0] = 1;
	state[1][1] = 1;
	//�������켣���򣬱�֤��һ���ȵڶ�����
	//���Ȱѹ켣���ڹ����ڴ���
	__shared__ SPoint queryTraS[MAXTHREAD];
	__shared__ SPoint traData[MAXTHREAD];
	if (threadID < lenT) {
		traData[threadID] = candidateTra[blockID][threadID];
	}
	if (threadID < queryLength) {
		queryTraS[threadID] = queryTra[threadID];
	}
	const SPoint *tra1, *tra2; //��֤tra1��tra2��
	int len1, len2;
	if (lenT >= queryLength) {
		tra1 = queryTraS;
		tra2 = traData;
		len1 = queryLength;
		len2 = lenT;
	}
	else
	{
		tra1 = traData;
		tra2 = queryTraS;
		len1 = lenT;
		len2 = queryLength;
	}

	int myState;
	for (int i = 0; i <= iterNum - 1; i++) {//��i��dp
		if (i < len1 - 1) {
			if (threadID <= i) {
				SPoint p1 = tra1[threadID];
				SPoint p2 = tra2[i - threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
					subcost = 0;
				}
				else
					subcost = 1;
				int state_ismatch = state[0][threadID] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID+1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_ismatch;
			}
		}
		else if (i > iterNum - len1) {
			if (threadID <= iterNum - i - 1) {
				SPoint p1 = tra1[threadID+len1-(iterNum-i)];
				SPoint p2 = tra2[len2-1-threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
					subcost = 0;
				}
				else
					subcost = 1;
				int state_ismatch = state[0][threadID+1] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID + 1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_ismatch;
			}
		}
		else
		{
			if (threadID < len1) {
				SPoint p1 = tra1[threadID];
				SPoint p2 = tra2[i-threadID]; //�������ڴ��Ǿۼ����ʵ���
				bool subcost;
				if (((p1.x - p2.x)*(p1.x - p2.x) + (p1.y - p2.y)*(p1.y - p2.y)) < EPSILON) {
					subcost = 0;
				}
				else
					subcost = 1;
				int state_ismatch = state[0][threadID] + subcost;
				int state_up = state[1][threadID] + 1;
				int state_left = state[1][threadID + 1] + 1;
				if (state_ismatch < state_up)
					myState = state_ismatch;
				else if (state_left < state_up)
					myState = state_left;
				else
					myState = state_ismatch;
			}
		}
		//дmyState��share�ڴ�,ckecked
		int startidx;
		//���Ƚ�������д��ȫ���ڴ棬ȫд
		//startidx�Ǿɵ�����Ӧ����ȫ���ڴ��е�ַ����i-2����
		//����Ӧд��ȫ���ڴ����ʼλ��

		if (i-2 < len1 - 2) {
			startidx = (i-2 + 2)*(i-2 + 3) / 2;
			if (threadID <= i) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}
		else if (i-2 >= iterNum - len1) {
			startidx = (len1 + 1)*(len2 + 1) - (iterNum - (i-2))*(iterNum - (i-2) + 1) / 2;
			if (threadID <= iterNum - i + 1 ) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}
		else
		{
			startidx = (len1 + 1)*((i - 2) - (len1 - 2)) + len1*(len1 + 1) / 2;
			if (threadID <= len1) {
				stateTableGPU[blockID][threadID + startidx] = state[0][threadID];
			}
		}

		//�ƶ������ݵ�������
		state[0][threadID] = state[1][threadID];
		//д��������
		if (i < len1-1) {
			if (threadID <= i)
				state[1][threadID + 1] = myState;
			if (threadID == 0) {
				state[1][0] = i + 2;
				state[1][i + 2] = i + 2;
			}
		}
		else if (i >= iterNum - len1) {
			if (threadID <= iterNum - i - 1)
				state[1][threadID] = myState;
		}
		else
		{
			if (threadID < len1)
				state[1][threadID + 1] = myState;
			if (threadID == 0) {
				state[1][0] = i + 2;
			}
		}
	}
	//�����������һ�μ���һ�����ɽ���0��ɵ�
	if (threadID == 0)
		result[blockID] = myState;
}

//�Ȱ����ܷ���һ��SMִ��һ��DP�����������ٷֱ��������kernel
//constructing...
int handleEDRdistance(const SPoint *queryTra, const SPoint **candidateTra, const int candidateNum, const int queryLength, const int *candidateLength,int *result) {
	int** stateTableGPU;
	//��GPU��Ϊ״̬������ڴ�
	hipMalloc((void**)&stateTableGPU, sizeof(int*)*candidateNum);
	for (int i = 0; i <= candidateNum - 1; i++) {
		hipMalloc((void**)&stateTableGPU[i], sizeof(int)*(candidateLength[i] + 1)*(queryLength + 1));
	}
	//���ͨ���������ķ������ݹ켣�����Ҫ��켣�����洢

}

