#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "cudaKernel.h"
#include "thrust\device_ptr.h"
#include "thrust\remove.h"
#include <stdio.h>
#include <assert.h>
#include <vector>
#include <iostream>
#include "WinTimer.h"
#include "ConstDefine.h"



#define CUDA_CALL(x) { const hipError_t a = (x); if (a!= hipSuccess) { printf("\nCUDA Error: %s(err_num=%d)\n", hipGetErrorString(a), a); hipDeviceReset(); assert(0);}}

#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line)
{
	hipError_t err = hipGetLastError();

	if (hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
			file, line, errorMessage, (int)err, hipGetErrorString(err));
			exit(EXIT_FAILURE);
	}
}


//using namespace thrust;
static const int THREAD_N = 256; //ÿ��block�߳���

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



#ifdef _CELL_BASED_STORAGE
int putCellDataSetIntoGPU(Point* pointsPtr, Point*& pointsPtrGPU, int pointNum) {
	CUDA_CALL(hipSetDevice(0));
	CUDA_CALL(hipMalloc((void**)&pointsPtrGPU, pointNum * sizeof(Point))); //�������ݵ��ڴ�
	//debug
	//std::cout << pointNum << std::endl;
	//debug
	CUDA_CALL(hipMemcpy(pointsPtrGPU, pointsPtr, pointNum * sizeof(Point), hipMemcpyHostToDevice));//���ݿ�����gpu��
	return 0;
}
__global__ void cudaRangeQuery(int* rangeStarts, int* rangeEnds, int candidateCellNum, const Point* pointsPtr, const float xmin, const float ymin, const float xmax, const float ymax, const int *resultOffset, Point* resultPtrCuda) {
	int cellNo = blockIdx.x; //candidate����ڼ���cell 0,1,2,....
	if (cellNo >= candidateCellNum) return;
	int tid = threadIdx.x;
	if (tid >= 256) return;
	int pointNum = rangeEnds[cellNo] - rangeStarts[cellNo] + 1;//blockҪ��������cell����ô�����
	const int offset = rangeStarts[cellNo];
	for (int i = tid; i <= pointNum - 1; i += THREAD_N) {
		float x = pointsPtr[offset + i].x;
		float y = pointsPtr[offset + i].y;
		uint32_t tid = pointsPtr[offset + i].tID;
		uint32_t time = pointsPtr[offset + i].time;
		if (x <= xmax &&x >= xmin&&y <= ymax&&y >= ymin) {
			resultPtrCuda[resultOffset[cellNo] + i].x = x;
			resultPtrCuda[resultOffset[cellNo] + i].y = y;
			resultPtrCuda[resultOffset[cellNo] + i].tID = tid;
			resultPtrCuda[resultOffset[cellNo] + i].time = time;
		}
		else
			resultPtrCuda[resultOffset[cellNo] + i].tID = -1;
	}
}
int cudaRangeQueryHandler(int* candidateCells, int* rangeStarts, int* rangeEnds, int candidateCellNum,float xmin, float ymin, float xmax, float ymax, Point*& resultsGPU, int& resultNum,Point *pointsPtrGPU,Point *&result) {
	//��һ��������ʱû���ã�ע������candidatecells[i]�Ѿ�������cell��id������ֻ��ǿ�
	//���ĸ�������ʾ�ǿյ�cell����
	//����candidate���еĵ��������gpu�ڿ�����ͬ��С�Ŀռ���flag��rangestart��rangeend����Ӧcandidatecell�ڵĲ�������AllPoints����ʼ�±����ֹ�±�
	//�����������͵����ڶ��������������һ���Ǳ�������GPU��ַ���ڶ����ǽ���ĸ���
	//PointsPtrGPU�����ݼ���gpu�ĵ�ַ
	//MyTimer timer1;
	//timer1.start();
	int counter = 0;
	int *resultOffset = (int*)malloc(sizeof(int)*candidateCellNum);
	//std::cout << candidateCellNum << ":"<<std::endl;
	for (int i = 0; i <= candidateCellNum - 1; i++) {
		resultOffset[i] = counter;
		////debug
		//std::cout << "(" << rangeStarts[i] << "," << rangeEnds[i] << ");"<<"["<<resultOffset[i]<<"]";
		////debug
		counter += rangeEnds[i] - rangeStarts[i] + 1;
	}
	int totalPointNumInCandidate = counter;


	int* candidateCellsCuda = NULL, *rangeStartsCuda = NULL, *rangeEndsCuda = NULL, *resultOffsetCuda = NULL;

	CUDA_CALL(hipMalloc((void**)&resultsGPU, sizeof(Point)*totalPointNumInCandidate));
	//��range��cell��Ϣд��gpu
	//CUDA_CALL(hipMalloc((void**)&candidateCellsCuda, sizeof(int)*candidateCellNum));
	CUDA_CALL(hipMalloc((void**)&rangeStartsCuda, candidateCellNum*sizeof(int)));
	//std::cout << "\n" << candidateCellNum*sizeof(int) << "\n";
	CUDA_CALL(hipMalloc((void**)&rangeEndsCuda, candidateCellNum*sizeof(int)));
	CUDA_CALL(hipMalloc((void**)&resultOffsetCuda, candidateCellNum*sizeof(int)));
	//CUDA_CALL(hipMemcpy(candidateCellsCuda, candidateCells, candidateCellNum*sizeof(int), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(rangeStartsCuda, rangeStarts, candidateCellNum*sizeof(int), hipMemcpyHostToDevice));

	CUDA_CALL(hipMemcpy(rangeEndsCuda, rangeEnds, candidateCellNum*sizeof(int), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(resultOffsetCuda, resultOffset, candidateCellNum*sizeof(int), hipMemcpyHostToDevice));
	////debug
	//CUDA_CALL(hipMemcpy( rangeStarts, rangeStartsCuda, candidateCellNum*sizeof(int), hipMemcpyDeviceToHost));
	//CUDA_CALL(hipMemcpy( rangeEnds, rangeEndsCuda, candidateCellNum*sizeof(int), hipMemcpyDeviceToHost));
	//CUDA_CALL(hipMemcpy( resultOffset, resultOffsetCuda, candidateCellNum*sizeof(int), hipMemcpyDeviceToHost));
	//for (int i = 0; i <= candidateCellNum - 1; i++) {
	//	//debug
	//	std::cout << "(" << rangeStarts[i] << "," << rangeEnds[i] << ");" << "[" << resultOffset[i] << "]";
	//	//debug
	//}
	//debug
	//timer1.stop();
	//std::cout << timer1.ticks() << std::endl;
	//timer1.start();
	//����kernel�����ĳ����������������Ӧλ��д����AllPoints�е��±꣬����д��-1
	//ÿ��cell�����һ��block
	cudaRangeQuery <<<candidateCellNum, THREAD_N >>>(rangeStartsCuda, rangeEndsCuda, candidateCellNum, pointsPtrGPU, xmin, ymin, xmax, ymax, resultOffsetCuda, resultsGPU);
	//kernel���ý��������������idxsGPU�У����������������ӦԪ����������AllPoints���±꣬��������ϣ�����Ϊ-1
	//CUDA_CALL(hipFree(candidateCellsCuda));
	CUDA_CALL(hipFree(rangeStartsCuda));
	CUDA_CALL(hipFree(rangeEndsCuda));
	CUDA_CALL(hipFree(resultOffsetCuda));
	//getLastCudaError("Error in Calling 'kernel'");
	//ʹ��Thrustɾ������-1���õ����ս��
	//timer1.stop();
	//std::cout << timer1.ticks() << std::endl;

	//���н���ϲ�
	//test

	//timer1.start();
	Point *resultset = NULL;
	resultset = (Point*)malloc(totalPointNumInCandidate*sizeof(Point));
	CUDA_CALL(hipMemcpy(resultset, resultsGPU, sizeof(Point)*totalPointNumInCandidate, hipMemcpyDeviceToHost));
	std::vector<Point> *resultPoint = new std::vector<Point>;
	for (int i = 0; i <= totalPointNumInCandidate - 1; i++) {
		if (resultset[i].tID != -1)
		{
			resultPoint->push_back(resultset[i]);
		}
	}
	result = &resultPoint->at(0);
	free(resultset);
	//test
	//timer1.stop();
	//std::cout << timer1.ticks() << std::endl;
	
	//���н���ϲ�
	
	//thrust::device_ptr<int> idxsPtr = thrust::device_pointer_cast(idxsGPU);
	//int a;
	//hipMemcpy(&a, idxsGPU, 1, hipMemcpyDeviceToHost);
	//size_t num = thrust::remove(idxsPtr, idxsPtr + totalPointNumInCandidate-1, -1) - idxsPtr;
	//int *result = (int*)malloc(sizeof(int)*num);
	//thrust::copy(idxsPtr, idxsPtr + num, result);
	//resultNum = num;
	//resultIdx = result;

	//CUDA_CALL(hipFree(idxsGPU));


	return 0;
}





#else
int cudaRangeQueryHandler(Point* pointsPtr, int pointNum, float xmin, float ymin, float xmax, float ymax, Point*& resultsPtr, int& resultNum) {
	Point* pointsPtrCuda = NULL;
	Point* resultPtrCuda = NULL;
	CUDA_CALL(hipMalloc((void**)&pointsPtrCuda, pointNum * sizeof(Point))); //�������ݵ��ڴ�
	CUDA_CALL(hipMalloc((void**)&resultPtrCuda, pointNum * sizeof(Point))); //gpu�ڴ洢����ĵط�
	CUDA_CALL(hipMemcpy(pointsPtrCuda, pointsPtr, pointNum * sizeof(Point), hipMemcpyHostToDevice));//���ݿ�����gpu��

																									  //���ú˺����������ݣ��������gpu��

																									  //ȡ�����ݣ�����
	return 0;
}
#endif


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
