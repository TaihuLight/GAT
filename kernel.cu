#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "cudaKernel.h"
#include "thrust/device_ptr.h"
#include "thrust/remove.h"
#include <stdio.h>
#include <assert.h>
#include <vector>
#include <iostream>
#include "ConstDefine.h"





inline void __getLastCudaError(const char *errorMessage, const char *file, const int line)
{
	hipError_t err = hipGetLastError();

	if (hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
			file, line, errorMessage, (int)err, hipGetErrorString(err));
			exit(EXIT_FAILURE);
	}
}


//using namespace thrust;
//static const int MAXTHREAD = 512; //ÿ��block�߳���

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void CUDAwarmUp() {
	CUDA_CALL(hipSetDeviceFlags(hipDeviceMapHost));
	CUDA_CALL(hipSetDevice(0));
	
}


#ifdef _CELL_BASED_STORAGE
int putCellDataSetIntoGPU(Point* pointsPtr, Point*& pointsPtrGPU, int pointNum) {
	
	CUDA_CALL(hipMalloc((void**)&pointsPtrGPU, pointNum * sizeof(Point))); //�������ݵ��ڴ�
	//debug
	//std::cout << pointNum << std::endl;
	//debug
	CUDA_CALL(hipMemcpy(pointsPtrGPU, pointsPtr, pointNum * sizeof(Point), hipMemcpyHostToDevice));//���ݿ�����gpu��
	return 0;
}
__global__ void cudaRangeQuery(int* rangeStarts, int* rangeEnds, int candidateCellNum, const Point* pointsPtr, const float xmin, const float ymin, const float xmax, const float ymax, const int *resultOffset, Point* resultPtrCuda) {
	int cellNo = blockIdx.x; //candidate����ڼ���cell 0,1,2,....
	if (cellNo >= candidateCellNum) return;
	int tid = threadIdx.x;
	if (tid >= 256) return;
	int pointNum = rangeEnds[cellNo] - rangeStarts[cellNo] + 1;//blockҪ��������cell����ô�����
	const int offset = rangeStarts[cellNo];
	for (int i = tid; i <= pointNum - 1; i += MAXTHREAD) {
		float x = pointsPtr[offset + i].x;
		float y = pointsPtr[offset + i].y;
		uint32_t tid = pointsPtr[offset + i].tID;
		uint32_t time = pointsPtr[offset + i].time;
		if (x <= xmax &&x >= xmin&&y <= ymax&&y >= ymin) {
			resultPtrCuda[resultOffset[cellNo] + i].x = x;
			resultPtrCuda[resultOffset[cellNo] + i].y = y;
			resultPtrCuda[resultOffset[cellNo] + i].tID = tid;
			resultPtrCuda[resultOffset[cellNo] + i].time = time;
		}
		else
			resultPtrCuda[resultOffset[cellNo] + i].tID = -1;
	}
}

__global__ void cudaRangeQueryTest(RangeQueryStateTable* stateTable, int stateTableLength, uint8_t* result, 
	const int maxTrajNum) {
	int bID = blockIdx.x;
	int tID = threadIdx.x;
	__shared__ RangeQueryStateTable sharedStateTable;
	// __shared__ uint8_t resultTemp[10000]; //10K
	sharedStateTable = (stateTable[bID]);
	int jobID = sharedStateTable.queryID;
	SPoint *baseAddr = (SPoint*)(sharedStateTable.ptr);
	int candidateNum = sharedStateTable.candidatePointNum;//��block����Ҫ��ѯ�ĵ�ĸ���
	//int resultOffset = bID*maxPointNumInStateTable; //��block�Ľ������ʼ��ַ
	for (int i = 0; i <= candidateNum / MAXTHREAD-1; i++) {
		SPoint p = *(baseAddr + (i*MAXTHREAD + tID));
		//result[i*MAXTHREAD + tID + resultOffset].idx = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) &&
			//(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))*(i*MAXTHREAD + tID);//�����֤ͨ�������ֵΪ�����ţ�����Ϊ0
		//result[i*MAXTHREAD + tID + resultOffset].jobID = bID;
		//result[resultOffset + (i*MAXTHREAD + tID)] = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && 
		//		(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin));
		if((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))
			result[jobID*maxTrajNum + p.tID] = 1;
		//�����֤ͨ��������Ӧλ����Ϊ1

		//__syncthreads();
	}
	if (tID < candidateNum - candidateNum / MAXTHREAD * MAXTHREAD) {
		SPoint p = *(baseAddr + (candidateNum / MAXTHREAD * MAXTHREAD + tID));
		//result[candidateNum / MAXTHREAD * MAXTHREAD + tID + resultOffset].idx = ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) &&
		//	(p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))*(candidateNum / MAXTHREAD * MAXTHREAD + tID);//�����֤ͨ�������ֵΪ�����ţ�����Ϊ0
		//result[candidateNum / MAXTHREAD * MAXTHREAD + tID + resultOffset].jobID = bID;
		//result[resultOffset + (candidateNum / MAXTHREAD * MAXTHREAD + tID)] = ((p.x<sharedStateTable.xmax) &&
		//	(p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin));
		if ((p.x<sharedStateTable.xmax) && (p.x>sharedStateTable.xmin) && (p.y<sharedStateTable.ymax) && (p.y>sharedStateTable.ymin))
			result[jobID*maxTrajNum + p.tID] = 1;
	}
	//else {
	//	//result[candidateNum / MAXTHREAD * MAXTHREAD + tID + resultOffset].idx = 0; //������Ĳ��֣�ֱ����Ϊ��Ч����
	//}
	//__syncthreads();
	//__syncthreads();
	//int globalTID = blockDim.x * blockIdx.x + threadIdx.x;
	//if (globalTID < stateTableLength) {

	//}
}

int cudaRangeQueryTestHandler(RangeQueryStateTable* stateTableGPU, int stateTableLength, uint8_t *result, int maxTrajNum
	, int maxJobNum, hipStream_t stream) {
	//RangeQueryResultGPU* resultGPU;
	//MyTimer timer;
	uint8_t* resultGPU;
	//int resultByteNum = (maxPointNum)/8+1;//ÿ�������Ҫ�ü���byte���棬���ܰ����أ�ֻ�ܰ��ֽ�
	CUDA_CALL(hipMalloc((void**)&resultGPU, (maxTrajNum)*maxJobNum));//selective��һ��
	CUDA_CALL(hipMemset(resultGPU, 0, (maxTrajNum)*maxJobNum));
	//timer.start();
	//�����һ���ڴ棬ÿ��stateTable��ռ�ݵ��ڴ������
	//CUDA_CALL(hipMalloc((void**)&resultGPU, (maxPointNum)*stateTableLength));
	
	//CUDA_CALL(hipMalloc((void**)&resultGPU, maxPointNum*stateTableLength*sizeof(RangeQueryResultGPU)));
	//timer.stop();
	//std::cout << "Time 1:" << timer.elapse() << "ms" << std::endl;

	//timer.start();	
	cudaRangeQueryTest <<<stateTableLength, MAXTHREAD,0, stream >>>(stateTableGPU, stateTableLength, resultGPU, maxTrajNum);
	CUDA_CALL(hipDeviceSynchronize());
	//timer.stop();
	//std::cout << "Time 2:" << timer.elapse() << "ms" << std::endl;

	//timer.start();
	
	CUDA_CALL(hipMemcpy(result, resultGPU, (maxTrajNum)*maxJobNum, hipMemcpyDeviceToHost));

	//timer.stop();
	//std::cout << "Time 3:" << timer.elapse() << "ms" << std::endl;
	return 0;
}

int cudaRangeQueryHandler(int* candidateCells, int* rangeStarts, int* rangeEnds, int candidateCellNum,float xmin, float ymin, float xmax, float ymax, Point*& resultsGPU, int& resultNum,Point *pointsPtrGPU,Point *&result) {
	//��һ��������ʱû���ã�ע������candidatecells[i]�Ѿ�������cell��id������ֻ��ǿ�
	//���ĸ�������ʾ�ǿյ�cell����
	//����candidate���еĵ��������gpu�ڿ�����ͬ��С�Ŀռ���flag��rangestart��rangeend����Ӧcandidatecell�ڵĲ�������AllPoints����ʼ�±����ֹ�±�
	//�����������͵����ڶ��������������һ���Ǳ�������GPU��ַ���ڶ����ǽ���ĸ���
	//PointsPtrGPU�����ݼ���gpu�ĵ�ַ
	//MyTimer timer1;
	//timer1.start();
	int counter = 0;
	int *resultOffset = (int*)malloc(sizeof(int)*candidateCellNum);
	//std::cout << candidateCellNum << ":"<<std::endl;
	for (int i = 0; i <= candidateCellNum - 1; i++) {
		resultOffset[i] = counter;
		////debug
		//std::cout << "(" << rangeStarts[i] << "," << rangeEnds[i] << ");"<<"["<<resultOffset[i]<<"]";
		////debug
		counter += rangeEnds[i] - rangeStarts[i] + 1;
	}
	int totalPointNumInCandidate = counter;


	int *rangeStartsCuda = NULL, *rangeEndsCuda = NULL, *resultOffsetCuda = NULL;

	CUDA_CALL(hipMalloc((void**)&resultsGPU, sizeof(Point)*totalPointNumInCandidate));
	//��range��cell��Ϣд��gpu
	//CUDA_CALL(hipMalloc((void**)&candidateCellsCuda, sizeof(int)*candidateCellNum));
	CUDA_CALL(hipMalloc((void**)&rangeStartsCuda, candidateCellNum*sizeof(int)));
	//std::cout << "\n" << candidateCellNum*sizeof(int) << "\n";
	CUDA_CALL(hipMalloc((void**)&rangeEndsCuda, candidateCellNum*sizeof(int)));
	CUDA_CALL(hipMalloc((void**)&resultOffsetCuda, candidateCellNum*sizeof(int)));
	//CUDA_CALL(hipMemcpy(candidateCellsCuda, candidateCells, candidateCellNum*sizeof(int), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(rangeStartsCuda, rangeStarts, candidateCellNum*sizeof(int), hipMemcpyHostToDevice));

	CUDA_CALL(hipMemcpy(rangeEndsCuda, rangeEnds, candidateCellNum*sizeof(int), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(resultOffsetCuda, resultOffset, candidateCellNum*sizeof(int), hipMemcpyHostToDevice));
	////debug
	//CUDA_CALL(hipMemcpy( rangeStarts, rangeStartsCuda, candidateCellNum*sizeof(int), hipMemcpyDeviceToHost));
	//CUDA_CALL(hipMemcpy( rangeEnds, rangeEndsCuda, candidateCellNum*sizeof(int), hipMemcpyDeviceToHost));
	//CUDA_CALL(hipMemcpy( resultOffset, resultOffsetCuda, candidateCellNum*sizeof(int), hipMemcpyDeviceToHost));
	//for (int i = 0; i <= candidateCellNum - 1; i++) {
	//	//debug
	//	std::cout << "(" << rangeStarts[i] << "," << rangeEnds[i] << ");" << "[" << resultOffset[i] << "]";
	//	//debug
	//}
	//debug
	//timer1.stop();
	//std::cout << timer1.ticks() << std::endl;
	//timer1.start();
	//����kernel�����ĳ����������������Ӧλ��д����AllPoints�е��±꣬����д��-1
	//ÿ��cell�����һ��block
	cudaRangeQuery <<<candidateCellNum, MAXTHREAD >>>(rangeStartsCuda, rangeEndsCuda, candidateCellNum, pointsPtrGPU, xmin, ymin, xmax, ymax, resultOffsetCuda, resultsGPU);
	//kernel���ý��������������idxsGPU�У����������������ӦԪ����������AllPoints���±꣬��������ϣ�����Ϊ-1
	//CUDA_CALL(hipFree(candidateCellsCuda));
	CUDA_CALL(hipFree(rangeStartsCuda));
	CUDA_CALL(hipFree(rangeEndsCuda));
	CUDA_CALL(hipFree(resultOffsetCuda));
	//getLastCudaError("Error in Calling 'kernel'");
	//ʹ��Thrustɾ������-1���õ����ս��
	//timer1.stop();
	//std::cout << timer1.ticks() << std::endl;

	//���н���ϲ�
	//test

	//timer1.start();
	Point *resultset = NULL;
	resultset = (Point*)malloc(totalPointNumInCandidate*sizeof(Point));
	CUDA_CALL(hipMemcpy(resultset, resultsGPU, sizeof(Point)*totalPointNumInCandidate, hipMemcpyDeviceToHost));
	std::vector<Point> *resultPoint = new std::vector<Point>;
	for (int i = 0; i <= totalPointNumInCandidate - 1; i++) {
		if (resultset[i].tID != -1)
		{
			resultPoint->push_back(resultset[i]);
		}
	}
	result = &resultPoint->at(0);
	free(resultset);
	//test
	//timer1.stop();
	//std::cout << timer1.ticks() << std::endl;
	
	//���н���ϲ�
	
	//thrust::device_ptr<int> idxsPtr = thrust::device_pointer_cast(idxsGPU);
	//int a;
	//hipMemcpy(&a, idxsGPU, 1, hipMemcpyDeviceToHost);
	//size_t num = thrust::remove(idxsPtr, idxsPtr + totalPointNumInCandidate-1, -1) - idxsPtr;
	//int *result = (int*)malloc(sizeof(int)*num);
	//thrust::copy(idxsPtr, idxsPtr + num, result);
	//resultNum = num;
	//resultIdx = result;

	//CUDA_CALL(hipFree(idxsGPU));


	return 0;
}





#else
int cudaRangeQueryHandler(Point* pointsPtr, int pointNum, float xmin, float ymin, float xmax, float ymax, Point*& resultsPtr, int& resultNum) {
	Point* pointsPtrCuda = NULL;
	Point* resultPtrCuda = NULL;
	CUDA_CALL(hipMalloc((void**)&pointsPtrCuda, pointNum * sizeof(Point))); //�������ݵ��ڴ�
	CUDA_CALL(hipMalloc((void**)&resultPtrCuda, pointNum * sizeof(Point))); //gpu�ڴ洢����ĵط�
	CUDA_CALL(hipMemcpy(pointsPtrCuda, pointsPtr, pointNum * sizeof(Point), hipMemcpyHostToDevice));//���ݿ�����gpu��

																									  //���ú˺����������ݣ��������gpu��

																									  //ȡ�����ݣ�����
	return 0;
}
#endif


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}

// Helper function for using CUDA to add vectors in parallel.